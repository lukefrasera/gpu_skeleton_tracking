/*
 * dijkstras-test.cu
 *
 *  Created on: Apr 20, 2015
 *      Author: luke
 */



#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <climits>
#include <stdint.h>
#include <ctime>

void CudaMallocErrorCheck(void** ptr, int size);
void DijkstrasSetupCuda(int *V, int *E, int *We, int *sigma, int *F, int *U, int num_v, int num_e);
void Extremas(int *V, int *E, int num_v, int num_e, int *extrema_vertex, int source_vertex);
void Initialize(int *V, int *E, int num_v, int num_e, int **dev_V, int **dev_E, int **dev_U, int **dev_F, int **dev_sigma, int source);
int Minimum(int *U, int *sigma, int *V, int *E, int num_v, int num_e, int *dev_dest, int *dev_src);
__global__ void InitializeGPU(int *V, int *E, int *U, int *F, int *sigma, int src, int size_v, int size_e);
__global__ void Relax(int *U, int *F, int *sigma, int *V, int *E, int num_v, int num_e);
__global__ void Update(int *U, int *F, int *sigma, int delta, int size);

__global__ void reduce(int *g_idata, int *g_odata, unsigned int n, int *U, int *sigma);
__global__ void reduce_fix(int *g_idata, int *g_odata, unsigned int n, unsigned int s_size, unsigned int loops, int *U, int *sigma);
uint32_t NearestPowerTwo(uint32_t N);
uint32_t NearestPowerBase(uint32_t N, uint32_t base, uint32_t &power);

// Generate V_a, E_a, Start_a, End_a, Weight_a
int main(int argc, char **argv) {
  // Initialize graph
  int V[]    = {0, 1, 5, 7, 9};
  int E[]    = {1, 0, 2, 3, 4, 1, 4, 1, 4, 1, 2, 3};
  int Sv[]   = {0, 1, 1, 1, 1, 2, 2, 3, 3, 4, 4, 4};
  int Ev[]   = {1, 0, 2, 3, 4, 1, 4, 1, 4, 1, 2, 3};
  int We[]   = {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};

  // Initialize Unsettled, Frontier, Sigma function
  int sigma[]= {0,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1};  // -1 = inf
  int F[]    = {1, 0, 0, 0, 0};
  int U[]    = {0, 1, 1, 1, 1};

  DijkstrasSetupCuda(V, E, We, sigma, F, U, 5, 12);
}

void DijkstrasSetupCuda(int *V, int *E, int *We, int *sigma, int *F, int *U, int num_v, int num_e) {
  int extrema_vertex;
  Extremas(V, E, num_v, num_e, &extrema_vertex, 0);
}

void Extremas(int *V, int *E, int num_v, int num_e, int *extrema_vertex, int source_vertex) {
  // Define Unsettled sigma and Frontier nodes
  int *dev_U, *dev_sigma, *dev_F, *dev_V, *dev_E, *dev_src, *dev_dest;
  int delta = 0;
  float elapsedTime=0;

  // Initialize reduce function mem
  CudaMallocErrorCheck((void**)&dev_src, num_v*sizeof(int));
  CudaMallocErrorCheck((void**)&dev_dest, num_v*sizeof(int));

  Initialize(V, E, num_v, num_e, &dev_V, &dev_E, &dev_U, &dev_F, &dev_sigma, source_vertex);

//  Relax<<<1, 5>>>(dev_U, dev_F, dev_sigma, dev_V, dev_E, num_v, num_e);
//  int test = Minimum(dev_U, dev_sigma, dev_V, dev_E, num_v, num_e, dev_dest, dev_src);
//  Update<<<1,5>>>(dev_U, dev_F, dev_sigma, test, num_v);
//  printf("Test: %d\n", test);
//
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);
  while (delta != INT_MAX) {
    Relax<<<1, 5>>>(dev_U, dev_F, dev_sigma, dev_V, dev_E, num_v, num_e);
    delta = Minimum(dev_U, dev_sigma, dev_V, dev_E, num_v, num_e, dev_dest, dev_src);
    Update<<<1, 5>>>(dev_U, dev_F, dev_sigma, delta, num_v);
  }
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsedTime, start, end);
  printf("Elapsed Time: %f\n", elapsedTime);
  hipEventDestroy(start);
  hipEventDestroy(end);

  int sigma[num_v];
//  int V_t[num_v];
//  int U_t[num_v];
  hipMemcpy(sigma, dev_sigma, num_v*sizeof(int), hipMemcpyDeviceToHost);
//  cudaMemcpy(V_t, dev_F, num_v*sizeof(int), cudaMemcpyDeviceToHost);
//  cudaMemcpy(U_t, dev_U, num_v*sizeof(int), cudaMemcpyDeviceToHost);
  for (int i = 0; i < num_v; ++i) {
    printf("Sigma[%d]    : %d\n", i, sigma[i]);
//    printf("Frontier[%d] : %d\n", i, V_t[i]);
//    printf("Unsettled[%d]: %d\n", i, U_t[i]);
  }
}

void Initialize(int *V, int *E, int num_v, int num_e, int **dev_V, int **dev_E, int **dev_U, int **dev_F, int **dev_sigma, int source) {
  // Allocate the device memory
  CudaMallocErrorCheck((void**)dev_V, num_v*sizeof(int));
  CudaMallocErrorCheck((void**)dev_E, num_e*sizeof(int));
  CudaMallocErrorCheck((void**)dev_U, num_v*sizeof(int));
  CudaMallocErrorCheck((void**)dev_F, num_v*sizeof(int));
  CudaMallocErrorCheck((void**)dev_sigma, num_v*sizeof(int));

  // copy graph to device
  hipMemcpy(*dev_V, V, num_v*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(*dev_E, E, num_e*sizeof(int), hipMemcpyHostToDevice);
  // initialize Frontier
  // Initialize Unselttled
  // Initialize Sigma distance function
  int threads_per_block, blocks_per_dim;
  blocks_per_dim = num_v / 1024 + 1;
  threads_per_block = num_v / blocks_per_dim;

  InitializeGPU<<<blocks_per_dim, threads_per_block>>>(*dev_V, *dev_E, *dev_U, *dev_F, *dev_sigma, source, num_e, num_v);
}

__global__ void InitializeGPU(int *V, int *E, int *U, int *F, int *sigma, int src, int size_v, int size_e) {
  int offset = blockDim.x * blockIdx.x + threadIdx.x;
  int U_t, F_t, sigma_t;

  if (offset < size_v) {
    U_t = 1;
    F_t = 0;
    sigma_t = INT_MAX - 1;

    if (offset == src) {
      U_t = 0;
      F_t = 1;
      sigma_t = 0;
    }
    U[offset] = U_t;
    F[offset] = F_t;
    sigma[offset] = sigma_t;
  }
}

__global__ void Relax(int *U, int *F, int *sigma, int *V, int *E, int num_v, int num_e) {
  int offset = blockDim.x * blockIdx.x + threadIdx.x;

  if (offset < num_v) {
    if (F[offset] == 1) {
      for (int i = V[offset]; i < V[offset+1] && i < num_e; ++i) {
        if (U[E[i]] == 1) {
          atomicMin(&sigma[E[i]], sigma[offset] + 1);
        }
      }
    }
  }
}

__global__ void Update(int *U, int *F, int *sigma, int delta, int size) {
  int offset = blockDim.x * blockIdx.x + threadIdx.x;
  if (offset < size){
    F[offset] = 0;
    if (U[offset] == 1 && sigma[offset] <= delta) {
      U[offset] = 0;
      F[offset] = 1;
    }
  }
}

int Minimum(int *U, int *sigma, int *V, int *E, int num_v, int num_e, int *dev_dest, int *dev_src) {
  uint32_t blocks    = (num_v+1) / 1024 + 1;
  uint32_t threads = (num_v+1) / blocks / 2;


  uint32_t loops;
  uint32_t n_multiple = NearestPowerBase(num_v, threads * blocks * 2, loops);
  uint32_t dev_dest_size = NearestPowerTwo(blocks*loops);

  uint32_t share = NearestPowerTwo(threads);
//  printf("Blocks: %d, Threads:%d\n", blocks, threads);
  reduce_fix<<<blocks, threads, share*sizeof(int)>>>(V, dev_dest, n_multiple,
      share, loops, U, sigma);
  // Recall GPU function: Assumption Destination is power of 2. calculate block
  //                      and threads for each call.
  // GPU Call loop until Threshold
  if (dev_dest_size > 1024) {
    threads = 512;
    blocks = dev_dest_size / threads / 2;
  } else {
    threads = dev_dest_size / 2;
    blocks = 1;
  }

  while (dev_dest_size > 1) {
    int * temp = dev_dest;
    dev_dest = dev_src;
    dev_src = temp;
    reduce<<<blocks, threads, threads*sizeof(int)>>>(dev_src, dev_dest,
      dev_dest_size, U, sigma);
    dev_dest_size = blocks;
    if (dev_dest_size > 1024) {
      threads = 512;
      blocks = dev_dest_size / threads / 2;
    } else {
      threads = dev_dest_size / 2;
      blocks = 1;
    }
  }
  int result;
  hipMemcpy(&result, dev_dest, sizeof(int), hipMemcpyDeviceToHost);
  return result;
}
void CudaMallocErrorCheck(void** ptr, int size) {
  hipError_t err = hipMalloc(ptr, size);
  if (err != hipSuccess) {
    printf("Error: %s", hipGetErrorString(err));
    exit(1);
  }
}

uint32_t NearestPowerTwo(uint32_t N) {
  uint32_t result = 1;
  while (result < N) {
    result <<= 1;
  }
  return result;
}

uint32_t NearestPowerBase(uint32_t N, uint32_t base, uint32_t &power) {
  uint32_t result = base;
  power = 1;
  while (result < N) {
    result += base;
    power++;
  }
  return result;
}

__global__ void reduce(int *g_idata, int *g_odata, unsigned int n, int *U, int *sigma) {
  // Pointer to shared memory
  extern __shared__ int share_mem[];
  unsigned int thread_id = threadIdx.x;
  unsigned int block_id = blockIdx.x;
  unsigned int block_dim = blockDim.x;
  unsigned int offset = block_id*block_dim*2 + thread_id;

  // Temp result float
  int result = (offset < n && U[offset] == 1) ? g_idata[offset] : INT_MAX;

  // Perform summation
  if (offset + block_dim < n && U[offset + block_dim] == 1)
    result = min(result, g_idata[offset+block_dim]);
  share_mem[thread_id] = result;
  // Sync Threads in a single Block
  __syncthreads();

  // store result to shared memory
  for (unsigned int s=block_dim/2; s>0; s>>=1) {
    if (thread_id < s) {
      share_mem[thread_id] = result = min(result, share_mem[thread_id + s]);
    }
    __syncthreads();
  }

  // Store result to output data pointer
  if (thread_id == 0) g_odata[block_id] = result;
}

__global__ void reduce_fix(int *g_idata, int *g_odata, unsigned int n, unsigned int s_size, unsigned int loops, int *U, int *sigma) {
  // Pointer to shared memory
  extern __shared__ int share_mem[];
  unsigned int thread_id = threadIdx.x;
  for (int i = 0; i < loops; ++i) {
    unsigned int offset = blockIdx.x*blockDim.x*2 + threadIdx.x + blockDim.x * 2 * gridDim.x * i;

    // Temp result float
    int result = (offset < n && U[offset] == 1) ? g_idata[offset] : INT_MAX;

    // Perform summation
    if (offset + blockDim.x < n && U[offset + blockDim.x] == 1)
      result = min(result, g_idata[offset+blockDim.x]);
    share_mem[thread_id] = result;
//    printf("Result: %d\n", result);
    // Sync Threads in a single Block
    int delta = s_size - blockDim.x;
    if (thread_id + delta > blockDim.x-1) {
      share_mem[thread_id+delta] = INT_MAX;
    }
    __syncthreads();

    // store result to shared memory
    for (unsigned int s=s_size/2; s>0; s>>=1) {
      if (thread_id < s) {
        share_mem[thread_id] = result = min(result, share_mem[thread_id + s]);
      }
      __syncthreads();
    }

    // Store result to output data pointer
    if (thread_id == 0) {
      g_odata[blockIdx.x+ gridDim.x*i] = result;
    }
  }
}
